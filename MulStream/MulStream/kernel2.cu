#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

//(A+B)/2=C
#define N (1024)
#define FULL (N*20)

__global__ void kernel(int* a, int* b, int* c)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N)
	{
		c[idx] = (a[idx] + b[idx]) / 2;
	}
}

int main(void)
{
	//��ѯ�豸����
	hipDeviceProp_t prop;
	int whichDevice;
	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);
	if (!prop.deviceOverlap)
	{
		printf("Device will not support overlap\n");
		return 0;
	}

	//��ʼ����ʱ���¼�
	hipEvent_t start, stop;
	float elapsedTime;
	//��������Bufferָ��
	hipStream_t stream0;
	hipStream_t stream1;
	int* host_a, * host_b, * host_c;
	int* dev_a0, * dev_b0, * dev_c0;
	int* dev_a1, * dev_b1, * dev_c1;

	//������ʱ��
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//��ʼ����
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	//��GPU������洢�ռ�
	hipMalloc((void**)&dev_a0, N * sizeof(int));
	hipMalloc((void**)&dev_b0, N * sizeof(int));
	hipMalloc((void**)&dev_c0, N * sizeof(int));
	hipMalloc((void**)&dev_a1, N * sizeof(int));
	hipMalloc((void**)&dev_b1, N * sizeof(int));
	hipMalloc((void**)&dev_c1, N * sizeof(int));
	//��CPU������洢�ռ䣬����ҳ�ڴ�
	hipHostAlloc((void**)&host_a, FULL * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_b, FULL * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_c, FULL * sizeof(int), hipHostMallocDefault);

	//��ʼ��A,B����
	for (int i = 0; i < FULL; i++)
	{
		host_a[i] = rand();
		host_b[i] = rand();
	}

	//��ʼ����
	hipEventRecord(start, 0);
	for (int i = 0; i < FULL; i += N*2)
	{
		//�����ݴ�CPU��ҳ�ڴ��д����GPU�Դ�
		hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		kernel << <N / 256, 256, 0, stream0 >> > (dev_a0, dev_b0, dev_c0);
		kernel << <N / 256, 256, 0, stream1 >> > (dev_a1, dev_b1, dev_c1);
		//����������GPU�Դ��д����CPU�ڴ���
		hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
		hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
	}
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time: %3.1f ms\n", elapsedTime);

	hipFree(dev_a0);
	hipFree(dev_b0);
	hipFree(dev_c0);
	hipFree(dev_a1);
	hipFree(dev_b1);
	hipFree(dev_c1);
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);
	return 0;
}
#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

//(A+B)/2=C
#define N (1024)
#define FULL (N*20)

__global__ void kernel(int* a, int* b, int* c)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < N)
	{
		c[idx] = (a[idx] + b[idx]) / 2;
	}
}

int run_with_1_stream()
{
	//初始化计时器事件
	hipEvent_t start, stop;
	float elapsedTime;
	//声明流和Buffer指针
	hipStream_t stream;
	int* host_a, * host_b, * host_c;
	int* dev_a, * dev_b, * dev_c;

	//创建计时器
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//初始化流
	hipStreamCreate(&stream);

	//在GPU端申请存储空间
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));
	//在CPU端申请存储空间，用锁页内存
	hipHostAlloc((void**)&host_a, FULL * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_b, FULL * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_c, FULL * sizeof(int), hipHostMallocDefault);

	//初始化A,B向量
	for (int i = 0; i < FULL; i++)
	{
		host_a[i] = rand();
		host_b[i] = rand();
	}

	//开始计算
	hipEventRecord(start, 0);
	for (int i = 0; i < FULL; i += N)
	{
		//将数据从CPU锁页内存中传输给GPU显存
		hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
		kernel << <N / 256, 256, 0, stream >> > (dev_a, dev_b, dev_c);
		//将计算结果从GPU显存中传输给CPU内存中
		hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
	}
	hipStreamSynchronize(stream);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("1 stream Time: %3.1f ms\n", elapsedTime);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipStreamDestroy(stream);
	return 0;
}

int run_with_2_stream()
{
	//初始化计时器事件
	hipEvent_t start, stop;
	float elapsedTime;
	//声明流和Buffer指针
	hipStream_t stream0;
	hipStream_t stream1;
	int* host_a, * host_b, * host_c;
	int* dev_a0, * dev_b0, * dev_c0;
	int* dev_a1, * dev_b1, * dev_c1;

	//创建计时器
	hipEventCreate(&start);
	hipEventCreate(&stop);
	//初始化流
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	//在GPU端申请存储空间
	hipMalloc((void**)&dev_a0, N * sizeof(int));
	hipMalloc((void**)&dev_b0, N * sizeof(int));
	hipMalloc((void**)&dev_c0, N * sizeof(int));
	hipMalloc((void**)&dev_a1, N * sizeof(int));
	hipMalloc((void**)&dev_b1, N * sizeof(int));
	hipMalloc((void**)&dev_c1, N * sizeof(int));
	//在CPU端申请存储空间，用锁页内存
	hipHostAlloc((void**)&host_a, FULL * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_b, FULL * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_c, FULL * sizeof(int), hipHostMallocDefault);

	//初始化A,B向量
	for (int i = 0; i < FULL; i++)
	{
		host_a[i] = rand();
		host_b[i] = rand();
	}

	//开始计算
	hipEventRecord(start, 0);
	for (int i = 0; i < FULL; i += N * 2)
	{
		//将数据从CPU锁页内存中传输给GPU显存
		hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		kernel << <N / 256, 256, 0, stream0 >> > (dev_a0, dev_b0, dev_c0);
		kernel << <N / 256, 256, 0, stream1 >> > (dev_a1, dev_b1, dev_c1);
		//将计算结果从GPU显存中传输给CPU内存中
		hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
		hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
	}
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("2 stream Time: %3.1f ms\n", elapsedTime);

	hipFree(dev_a0);
	hipFree(dev_b0);
	hipFree(dev_c0);
	hipFree(dev_a1);
	hipFree(dev_b1);
	hipFree(dev_c1);
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);
	return 0;
}


int main(void)
{
	//查询设备属性
	hipDeviceProp_t prop;
	int whichDevice;
	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);
	if (!prop.deviceOverlap)
	{
		printf("Device will not support overlap\n");
		return 0;
	}

	for (int i = 0; i < 1; i++)
	{
		//run_with_1_stream();
		run_with_2_stream();
	}

	return 0;
}
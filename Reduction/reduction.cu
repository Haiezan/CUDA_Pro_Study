#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <conio.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""


// CPU串行计算函数
float CPUsum(const float* x, const int N)
{
	float sum = 0;
	for (int i = 0; i < N; i++)
	{
		sum += x[i];
	}
	return sum;
}

// GPU单核穿行计算函数
__global__ void GPUsum(const float* x, float* sum, const int* N)
{
	*sum = 0;
	for (int i = 0; i < *N; i++)
	{
		*sum += x[i];
	}
}

#define ARRAY_SIZE 1<<20
#define ARRAY_SIZE_IN_BYTES (sizeof(float) * (ARRAY_SIZE))

int main(void)
{
	// 计时变量
	clock_t start, end;

	// 初始化数据
	int N = ARRAY_SIZE;
	float* x = new float[ARRAY_SIZE];
	float sum;
	for (int i = 0; i < ARRAY_SIZE; i++)
	{
		x[i] = 1;
	}
	//初始化GPU内存
	float* dx;
	float* dsum;
	int* dN;
	hipMalloc((void**)&dx, ARRAY_SIZE_IN_BYTES);
	hipMalloc((void**)&dsum, sizeof(float));
	hipMalloc((void**)&dN, sizeof(int));
	hipMemcpy(dx, x, ARRAY_SIZE_IN_BYTES, hipMemcpyHostToDevice);
	hipMemcpy(dN, &N, sizeof(int), hipMemcpyHostToDevice);


	/*CPU串行*/
	start = clock();
	// 运行CPU串行函数
	sum = CPUsum(x, ARRAY_SIZE);
	end = clock();

	// 输出CPU运行结果
	printf("Calculate sum by CPU: sum = %f\nElasped time: %ldms\n", sum, (end - start));
	printf("\n");

	/*GPU单核串行*/
	
	// 运行GPU串行函数
	start = clock();
	GPUsum << <1, 1 >> > (dx, dsum, dN);

	// 拷贝计算结果到CPU
	hipMemcpy(&sum, dsum, sizeof(int), hipMemcpyDeviceToHost);
	end = clock();

	// 输出GPU串行计算结果
	printf("Calculate sum by GPU: sum = %f\nElasped time: %ldms\n", sum, (end - start));
	printf("\n");





	hipFree(dx);
	hipFree(dsum);
	hipFree(dN);
	return 0;
}
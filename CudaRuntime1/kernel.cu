#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include<iostream>
#include <time.h>
using namespace std;

__global__ void kernel()
{
	printf("blockIdx = %d, threadIdx = %d\n", blockIdx.x, threadIdx.x);
	//__syncthreads();
}

int main()
{
	int* dn;
	int n = 0;
	hipMalloc((void**)&dn, sizeof(int));
	hipMemcpy(dn, &n, sizeof(int), hipMemcpyHostToDevice);

	kernel <<<1, 256 >>> ();
	hipDeviceSynchronize();
	system("pause");
	return 0;
}